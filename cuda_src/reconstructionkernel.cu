#include "hip/hip_runtime.h"
#include "deviceVars.h"
#include "reconstruction.h"


#define MASK_W 7
#define MASK_H 7
#define TILE_W 16 //It has to be same size as block
#define TILE_H 16 //It has to be same size as block
#define MASK_R (MASK_W / 2)

#define w (TILE_W + MASK_W -1)
#define clamp(x) (min(max((x), 0.0), 1.0))
#define ThreadPerBlock TILE_H*TILE_W
#define PAD 3


hipStream_t R, G, B;
hipEvent_t Recon_start, Recon_end;
float recon_time;

__device__ void StSPk_Operation(float *d_Vector, float *d_x, int *pattern);
__device__ void additionScalar(float *d_Vector, float *d_first, float *d_second, float scalar);
__device__ void multiplyA(float *d_Vector, float *device_x, float *d_x, int *pattern, float *convResult, int dataH, int dataW, float *temp, float *holoArray);
__device__ void dotProduct(volatile float *cache, float *temp);
__device__ void convolve(float *data, float *temp, float *convResult, float *holoArray, int dataH, int dataW);




__device__ __constant__ float MASK[MASK_W * MASK_H];

__global__ void reconstructionKernel(float *data, float *result, int *pattern, int dataH, int dataW, volatile float *device_x, volatile float *device_p)
{

    __shared__ float holoArray[w*w]; //contains holo elements
    __shared__ float holoArrayNull[w*w];
//    __shared__ float holoTemp[w*w]; //contains temporary data for convolution
	__shared__ float temp[ThreadPerBlock];
    __shared__ float convResult[ThreadPerBlock];
    __shared__ float d_Vector[ThreadPerBlock];
    __shared__ float d_current_x[ThreadPerBlock];
    __shared__ float d_current_r[ThreadPerBlock];
    __shared__ float d_current_p[ThreadPerBlock];
    __shared__ float d_next_x[ThreadPerBlock];
    __shared__ float d_next_r[ThreadPerBlock];
    __shared__ float d_next_p[ThreadPerBlock];
    volatile __shared__ float cache_crnt_r[ThreadPerBlock]; //for dot product only
    volatile __shared__ float cache_crnt_p[ThreadPerBlock]; //for dot product only
    volatile __shared__ float cache_next_r[ThreadPerBlock]; //for dot product only
    __shared__ float cache[ThreadPerBlock];
    __shared__ int pixels[ThreadPerBlock];

    float dot_Num;
    float dot_Denom;
    float dot_alpha;
    float dot_beta;

    int GW = gridDim.x * blockDim.x + (gridDim.x + 1) * PAD;
    int GH = gridDim.y * blockDim.y + (gridDim.y + 1) * PAD;
    int STRIPSIZE = GW * (blockDim.y + PAD);


    float flag = 1.0f;
    int counter=0;

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = x + y * dataW;
    int bid = blockIdx.x + blockIdx.y * gridDim.x;
    int tx = x + (blockIdx.x +1) * PAD;
	int ty = y + (blockIdx.y +1)* PAD;

	int localIndex = threadIdx.x + threadIdx.y * TILE_W;
//	int holoIndex = tx + ty * dataW;
	int haloIndex = (blockIdx.y * STRIPSIZE) + (PAD * GW) + (threadIdx.y * GW) + (blockIdx.x + 1) * PAD + blockIdx.x * blockDim.x + threadIdx.x;
//	result[index] = data[index];

	__syncthreads();

	if(localIndex == 0)
	{
		int holoCounter = 0;
		int corner_x = blockIdx.x * blockDim.x + (blockIdx.x+1)* PAD;
		int corner_y = blockIdx.y * blockDim.y + (blockIdx.y+1)* PAD;
		for(int j= (corner_y - PAD); j<(corner_y+PAD+blockDim.y); j++)
		{
			for(int i = (corner_x - PAD); i<(corner_x+PAD+blockDim.x); i++)
			{
				int imageId = i + dataW * j;
				holoArray[holoCounter] = data[imageId];
				holoCounter++;
			}
		}
	}

	__syncthreads();
	if(localIndex == 1)
	{
		int holoCounter = 0;
		int corner_x = blockIdx.x * blockDim.x + (blockIdx.x+1)* PAD;
		int corner_y = blockIdx.y * blockDim.y + (blockIdx.y+1)* PAD;
		for(int j= (corner_y - PAD); j<(corner_y+PAD+blockDim.y); j++)
		{
			for(int i = (corner_x - PAD); i<(corner_x+PAD+blockDim.x); i++)
			{
//				int imageId = i + dataW * j;
				holoArrayNull[holoCounter] = 0.0f;
				holoCounter++;
			}
		}
	}
	__syncthreads();


	d_current_x[localIndex] = device_x[haloIndex];//data[index];
	cache[localIndex] = data[haloIndex];
	pixels[localIndex] = pattern[haloIndex];
	__syncthreads();
	multiplyA(d_Vector, d_current_x, d_current_x, pixels, convResult, dataH, dataW,temp, holoArray);

	__syncthreads();

	additionScalar(d_current_r, cache, d_Vector, -1); //cache = d_b; r = b - Ax
	d_current_p[localIndex] = d_current_r[localIndex];
	device_p[haloIndex] = d_current_p[localIndex];


	__syncthreads();

	// (fabs(flag - 0.00) > 1e-2) (fabs(flag - 0.00) > 1e-6) && (counter < 3) && (counter < 50)    fabs(flag - 0.00) > 1e-6


	while (counter < 15) //fabs(flag - 0.00) > 1e-6			counter < 50
		{
			//Dot product goes here and the answer will be stored in dot_result_num
			cache_crnt_r[localIndex] = d_current_r[localIndex]*d_current_r[localIndex];
			__syncthreads();

			dotProduct(cache_crnt_r, &dot_Num);
			__syncthreads();

			multiplyA(d_Vector, d_current_p, d_current_p, pixels,convResult,dataH, dataW, temp, holoArrayNull);
			__syncthreads();

			cache_crnt_p[localIndex] = d_current_p[localIndex] * d_Vector[localIndex];
			__syncthreads();
			dotProduct(cache_crnt_p, &dot_Denom);
			__syncthreads();
			dot_alpha = dot_Num / dot_Denom;
			additionScalar(d_next_x, d_current_x, d_current_p, dot_alpha);
			additionScalar(d_next_r, d_current_r,d_Vector, (-1)* dot_alpha);
			cache_next_r[localIndex] = d_next_r[localIndex] * d_next_r[localIndex];
			__syncthreads();
			dotProduct(cache_next_r, &dot_Denom); //beta = next_r/current_r
			__syncthreads();
			flag = sqrtf(dot_Denom);
			dot_beta = dot_Denom / dot_Num;
			additionScalar(d_next_p, d_next_r,d_current_p, dot_beta);
			d_current_r[localIndex] = d_next_r[localIndex];
			d_current_p[localIndex] = d_next_p[localIndex];
			d_current_x[localIndex] = d_next_x[localIndex];
			counter++;
			__syncthreads();
		}



	result[haloIndex] = d_next_x[localIndex];



}

__device__ void convolve(float *data, float *temp, float *convResult, float *holoArray, int dataH, int dataW)
{
	int localIndex = threadIdx.x + threadIdx.y * TILE_W;


    temp[localIndex] = data[localIndex];
    __syncthreads();

    float out = 0.0f;

    int corner_x = threadIdx.x - MASK_W/2;
    int corner_y = threadIdx.y - MASK_H/2;
    for(int y = 0; y<MASK_H; y++)
    {
   	 for(int x = 0; x<MASK_W; x++)
   	 {
   		 int i = corner_x + x;
   		 int j = corner_y + y;
   		 int maskIndex = x + y * MASK_W;
   		 int imageIndex;
   		 int holoIndex;
   		 float imageData;
   		 if(i<0 || i>=blockDim.x || j<0 || j>=blockDim.y)
   		 {
   			 i+=MASK_W/2;
   			 j+=MASK_H/2;
   			 holoIndex = i + j * (blockDim.x + 2 * PAD);
   			 imageData = holoArray[holoIndex];
   		 }
   		 else
   		 {
   			 imageIndex = i + j * blockDim.x;
   			 imageData = temp[imageIndex];
   		 }
   		 out += MASK[maskIndex] * imageData;
   	 }

    }
    convResult[localIndex] = out; //writing convolution result in shared memory for that block;
}

__device__ void multiplyA(float *d_Vector, float *device_x, float *d_x, int *pattern, float *convResult, int dataH, int dataW, float *temp, float *holoArray)
{
    convolve(device_x, temp, convResult, holoArray, dataH, dataW); //result will be also written on shared memory convResult;
    StSPk_Operation(d_Vector, d_x, pattern); //result will be also stored on temp shared memory
    additionScalar(d_Vector,d_Vector,convResult,1.0f); //result will be stored in result
}

__device__ void StSPk_Operation(float *d_Vector, float *d_x, int *pattern)
{
    int localIndex = threadIdx.x + threadIdx.y * TILE_W;
    d_Vector[localIndex] = d_x[localIndex] * pattern[localIndex];
}

__device__ void additionScalar(float *d_Vector, float *d_first, float *d_second, float scalar)
{
    int localIndex = threadIdx.x + threadIdx.y * TILE_W;
    d_Vector[localIndex] = d_first[localIndex] + scalar*d_second[localIndex];
}

 __device__ void dotProduct(volatile float *cache, float *temp)
{

    int localIndex = threadIdx.x + threadIdx.y * blockDim.x;

    if( localIndex < 128) {
    	cache[localIndex] += cache[localIndex + 128];
    }
    __syncthreads();
    if( localIndex < 64) {
        	cache[localIndex] += cache[localIndex + 64];
    }
    __syncthreads();
    if( localIndex < 32) {
    	cache[localIndex]+=cache[localIndex+32];
    	cache[localIndex]+=cache[localIndex+16];
    	cache[localIndex]+=cache[localIndex+8];
    	cache[localIndex]+=cache[localIndex+4];
    	cache[localIndex]+=cache[localIndex+2];
    	cache[localIndex]+=cache[localIndex+1];
    }

    __syncthreads();
    temp[0] = cache[0];

}

void initializeConvolutionFilter(float *kernel, int kernelLength)
{
	if(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, kernelLength * sizeof(float)) != hipSuccess)
	 {
		 printf("Copy to constant memory error\n");
	 }
	else printf("copy to MASK successful\n");


}
 void reconstructionFunction(dim3 grid, dim3 block, float *red, float *green, float *blue,
 		int *pattern, float *res_red, float *res_green, float *res_blue, int dataH, int dataW, float *device_x, float *device_p)
 {
/*
	 hipEventCreate(&Recon_start);
	 hipEventRecord(Recon_start, 0);
	 */
	hipStreamCreate(&R);
	hipStreamCreate(&G);
	hipStreamCreate(&B);

     reconstructionKernel<<<grid,block,0,R>>>(red, res_red, pattern, dataH, dataW, device_x, device_p);
     reconstructionKernel<<<grid,block,0,G>>>(green, res_green, pattern, dataH, dataW, device_x, device_p);
     reconstructionKernel<<<grid,block,0,B>>>(blue, res_blue, pattern, dataH, dataW, device_x, device_p);

     hipStreamDestroy(R);
     hipStreamDestroy(G);
     hipStreamDestroy(B);
/*
	 reconstructionKernel<<<grid,block>>>(red, res_red, pattern, dataH, dataW, device_x, device_p);
	      reconstructionKernel<<<grid,block>>>(green, res_green, pattern, dataH, dataW, device_x, device_p);
	      reconstructionKernel<<<grid,block>>>(blue, res_blue, pattern, dataH, dataW, device_x, device_p);
	      */
     hipDeviceSynchronize();

     getLastCudaError("kernel failed for reconstruction\n");


 }
