#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "helper_functions.h"
#include "CI/memcpy.cu"
#include "CI/cubicPrefilter3D.cu"
#include "CI/cubicTex3D.cu"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "deviceVars.h"
#include "reconstruction.h"

#define PAD 3

hipEvent_t start, stop;
float volumeTime;

typedef unsigned int  uint;
typedef unsigned char uchar;
typedef unsigned short ushort;

hipArray *d_volumeArray = 0;
hipArray *volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>         transferTexIso;
texture<uchar, 3, hipReadModeNormalizedFloat> tex_cubic;
//texture<ushort, 3, hipReadModeNormalizedFloat> tex_cubic;
texture<float, 3, hipReadModeElementType> coeffs;


typedef struct
{
	float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
	float3 o;   // origin
	float3 d;   // direction
};

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__ float3 mul(const float3x4 &M, const float3 &v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

// transform vector by matrix with translation
__device__ float4 mul(const float3x4 &M, const float4 &v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}


void setTextureFilterMode(bool bLinearFilter)
{
	tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

void initCuda(void *h_volume, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	//	hipPitchedPtr d_volumeMem;
	//	size_t size = d_volumeMem.pitch * volumeSize.height * volumeSize.depth;
	//	h_volume = (VolumeType*)malloc(size);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates //hipAddressModeClamp //hipAddressModeBorder
	tex.addressMode[1] = hipAddressModeBorder;
	tex.addressMode[2] = hipAddressModeBorder;
	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
	/*
    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };
	 */
	float4 transferFunc[] =
	{
			{0.231372549,	0.298039216,	0.752941176,	0,},
			{0.266666667,	0.352941176,	0.8,	0.03125,},
			{0.301960784,	0.407843137,	0.843137255,	0.0625,},
			{0.341176471,	0.458823529,	0.882352941,	0.09375,},
			{0.384313725,	0.509803922,	0.917647059,	0.125,},
			{0.423529412,	0.556862745,	0.945098039,	0.15625,},
			{0.466666667,	0.603921569,	0.968627451,	0.1875,},
			{0.509803922,	0.647058824,	0.984313725,	0.21875,},
			{0.552941176,	0.690196078,	0.996078431,	0.25,},
			{0.596078431,	0.725490196,	1,	0.28125,},
			{0.639215686,	0.760784314,	1,	0.3125,},
			{0.682352941,	0.788235294,	0.992156863,	0.34375,},
			{0.721568627,	0.815686275,	0.976470588,	0.375,},
			{0.760784314,	0.835294118,	0.956862745,	0.40625,},
			{0.800000000,	0.850980392,	0.933333333,	0.4375,},
			{0.835294118,	0.858823529,	0.901960784,	0.46875,},
			{0.866666667,	0.866666667,	0.866666667,	0.5,},
			{0.898039216,	0.847058824,	0.819607843,	0.53125,},
			{0.925490196,	0.827450980,	0.772549020,	0.5625,},
			{0.945098039,	0.8,	0.725490196,	0.59375,},
			{0.960784314,	0.768627451,	0.678431373,	0.625,},
			{0.968627451,	0.733333333,	0.62745098,	0.65625,},
			{0.968627451,	0.694117647,	0.580392157,	0.6875,},
			{0.968627451,	0.650980392,	0.529411765,	0.71875,},
			{0.956862745,	0.603921569,	0.482352941,	0.75,},
			{0.945098039,	0.552941176,	0.435294118,	0.78125,},
			{0.925490196,	0.498039216,	0.388235294,	0.8125,},
			{0.898039216,	0.439215686,	0.345098039,	0.84375,},
			{0.870588235,	0.376470588,	0.301960784,	0.875,},
			{0.835294118,	0.31372549,	0.258823529,	0.90625,},
			{0.796078431,	0.243137255,	0.219607843,	0.9375,},
			{0.752941176,	0.156862745,	0.184313725,	0.96875,},
			{0.705882353,	0.015686275,	0.149019608,	1,}
	};
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArray;
	checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

	transferTex.filterMode = hipFilterModeLinear;
	transferTex.normalized = true;    // access with normalized texture coordinates
	transferTex.addressMode[0] = hipAddressModeBorder;//hipAddressModeClamp;   // wrap texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));

	//Creating TransferTexIso
	float4 transferFuncIso[] =
	{
			{  0.0, 1.0, 0.0, 1.0 },
			{  0.0, 1.0, 0.0, 1.0 }
	};

	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArrayIso;
	checkCudaErrors(hipMallocArray(&d_transferFuncArrayIso, &channelDesc3, sizeof(transferFuncIso)/sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayIso, 0, 0, transferFuncIso, sizeof(transferFuncIso), hipMemcpyHostToDevice));

	transferTexIso.filterMode = hipFilterModeLinear;
	transferTexIso.normalized = true;    // access with normalized texture coordinates
	transferTexIso.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(transferTexIso, d_transferFuncArrayIso, channelDesc3));





}

void freeCudaBuffers()
{
	checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_transferFuncArray));
}
//void initCudaCubicSurface(const ushort* voxels, uint3 volumeSize)
void initCudaCubicSurface(const uchar* voxels, uint3 volumeSize)
{

	// calculate the b-spline coefficients
	hipPitchedPtr bsplineCoeffs = CastVolumeHostToDevice(voxels, volumeSize.x, volumeSize.y, volumeSize.z);
	CubicBSplinePrefilter3DTimer((float*)bsplineCoeffs.ptr, (uint)bsplineCoeffs.pitch, volumeSize.x, volumeSize.y, volumeSize.z);

	// create the b-spline coefficients texture
	hipArray *coeffArray = 0;
	hipExtent volumeExtent = make_hipExtent(volumeSize.x, volumeSize.y, volumeSize.z);
	CreateTextureFromVolume(&coeffs, &coeffArray, bsplineCoeffs, volumeExtent, true);
	//    CUDA_SAFE_CALL(hipFree(bsplineCoeffs.ptr));  //they are now in the coeffs texture, we do not need this anymore
	hipFree(bsplineCoeffs.ptr);
	// Now create a texture with the original sample values for nearest neighbor and linear interpolation
	// Note that if you are going to do cubic interpolation only, you can remove the following code

	CreateTextureFromVolume(&tex_cubic, &volumeArray, voxels, volumeExtent, false);
	tex_cubic.addressMode[0] = hipAddressModeBorder;
	tex_cubic.addressMode[1] = hipAddressModeBorder;
	tex_cubic.addressMode[2] = hipAddressModeBorder;




}

__device__ float max( float value )
{
	if( value < 0.0 )
		return 0.0;
	else
		return value;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = __saturatef(rgba.w);
	return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ float4 bisection(float3 start, float3 next,float3 direction, float stepSize, float isoValue)
{
	float tstep = stepSize/2;
	float3 a= start;
	float3 b = start+direction*tstep;
	float3 c = next;
	float3 point;
	float val = 0.0f;
	float temp_a = tex3D(tex, a.x , a.y , a.z ) - isoValue;
	float temp_b = tex3D(tex, b.x , b.y , b.z ) - isoValue;
	float temp_c = tex3D(tex, c.x , c.y , c.z ) - isoValue;
	int count = 0;
	float4 sample = make_float4(0.0f);

	while(count<25)
	{

		if(fabs(temp_b) <= (1e-6))
		{
			break;
		}

		if(temp_a*temp_b < 0)
		{
			tstep = tstep/2;
			c = b;
			b = a + direction * tstep;
		}
		else if(temp_b * temp_c < 0)
		{
			a = b;
			tstep = (3/4)*stepSize;
			b = a + direction*tstep;
		}
		val = tex3D(tex, b.x , b.y , b.z );
		point = b;
		if(fabs(val - isoValue)<= (1e-6))
		{
			break;
		}
		count++;
	}

	/*
         while(count<25)
    {
        if(fabs(temp_b) <= (1e-6))
        {
            break;
        }
        if(temp_a*temp_b < 0)
        {
            tstep = tstep/2;
            c = b;
            b = a + direction * tstep;
        }
        else if(temp_b * temp_c < 0)
        {
            a = b;
            tstep = (3/4)*stepSize;
            b = a + direction*tstep;
        }
        val = tex3D(tex, b.x , b.y , b.z );
        point = b;
        if(fabs(val - isoValue)<= (1e-6))
        {
            break;
        }
        count++;
    }
	 */
	sample.w = val;
	sample.x = b.x;
	sample.y = b.y;
	sample.z = b.z;

	return sample;
}



__global__ void d_render(int *d_pattern, int *linPattern, int *d_xPattern, int *d_yPattern, float *d_vol, float *d_red, float *d_green, float *d_blue, float *res_red, float *res_green, float *res_blue, int imageW, int imageH,
		float density, float brightness,float transferOffset, float transferScale, bool isoSurface, float isoValue, bool lightingCondition, bool isoLinear, float tstep,bool cubic, bool cubicLight, bool superSample, int filterMethod, float *d_temp)
{
	int maxSteps =1000;

	//    const float tstep = 0.001f;
	const float opacityThreshold = 1.00f;
	float powerFactor = 32.0f;

	float4 backGround = make_float4(1.0f);
	float4 finalSum = make_float4(0.0f);
	float4 averageSum = make_float4(0.0f);
	float4 sum, col;
	float I = 5.5f;
	float ka = 0.25f; //0.0025f;
	float I_amb = 0.2;
	float kd = 0.7;
	float I_dif;
	float ks = 0.5;
	float I_spec;
	float phong = 0.0f;
	float tstepGrad = 0.001f;
	float4 value;
	float sample;


	float x_space, y_space, z_space, x_dim, y_dim, z_dim, xAspect, yAspect, zAspect;
	x_dim = d_vol[0];
	y_dim = d_vol[1];
	z_dim = d_vol[2];

	x_space = d_vol[3];
	y_space = d_vol[4];
	z_space = d_vol[5];

	int pixel = (int)d_vol[6];
	//	brightness *= float(noRays);

	xAspect = (((x_dim - 1) * x_space)/((x_dim - 1) * x_space));
	xAspect = (((y_dim - 1) * y_space)/((x_dim - 1) * x_space));
	xAspect = (((z_dim - 1) * z_space)/((x_dim - 1) * x_space));

	//	float3 minB = (make_float3(-x_dim/x_dim, -y_dim/x_dim, -z_dim/x_dim));
	//	float3 maxB = (make_float3(x_dim/x_dim, y_dim/x_dim, z_dim/x_dim));

	float3 minB = (make_float3(-x_space, -y_space, -z_space));
	float3 maxB = (make_float3(x_space, y_space, z_space));

	const float3 boxMin = minB;//make_float3(-0.9316f, -0.9316f, -0.5f);
	const float3 boxMax = maxB;//make_float3( 0.9316f, 0.9316f, 0.5f);

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	//    int y = blockIdx.y*blockDim.y + threadIdx.y;
	//	int id = blockIdx.x*blockDim.x + threadIdx.x;
	//	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int id = x;// + y * imageW;
	float displaceX = 1.0/float(imageW);
	float displaceY = 1.0/float(imageH);

	int xArray[5] = {0,-1,1,1,-1};
	int yArray[5] = {0,1,1,-1,-1};


	if(id>=pixel)
		return;

	int tempLin = linPattern[id];


	float u = (d_xPattern[id]/(float)imageW)*2.0f - 1.0f;
	float v = (d_yPattern[id]/(float)imageH)*2.0f - 1.0f;

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));


	int noRays;
	if(superSample)
	{
		noRays = 5;
	}
	else
	{
		noRays = 1;
	}
	float4 accumulatedSum[5];
	for(int i=0; i<noRays; i++)
	{
		accumulatedSum[i]= make_float4(0.0f);
	}
	for(int i= 0; i<noRays; i++)
	{
		sum = make_float4(0.0f);
		int tempX = xArray[i];
		int tempY = yArray[i];

		eyeRay.d = normalize(make_float3(u+(displaceX*tempX)/4.0f, v+(displaceY*tempY)/4.0f, -1.0f));
		eyeRay.d = normalize(make_float3(u+(displaceX*tempX)/2.0f, v+(displaceY*tempY)/2.0f, -1.0f));
		eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

		float tnear, tfar;
		int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);


		if (!hit)
		{
			d_red[tempLin] = backGround.x;
			res_red[tempLin] = backGround.x;
			d_green[tempLin] = backGround.y;
			res_green[tempLin] = backGround.y;
			d_blue[tempLin] = backGround.z;
			res_blue[tempLin] = backGround.z;

			return;

		}
		else
		{

			float grad_x, grad_y, grad_z;


			if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
			// march along ray from front to back, accumulating color
			float t = tnear;
			float3 pos = eyeRay.o + eyeRay.d*tnear;
			float3 step = eyeRay.d*tstep;
			col = make_float4(0.0f);
			sample = 0.0f;
			float3 next;
			float3 start, mid, end, gradPos;
			float preValue, postValue;


			bool flag = false;

			pos.x = (pos.x/x_space) * 0.5f + 0.5f;
			pos.y = (pos.y/y_space) * 0.5f + 0.5f;
			pos.z = (pos.z/z_space) * 0.5f + 0.5f;

			for (int i=0; i<maxSteps; i++)
			{
				if(lightingCondition)
				{
					isoSurface = false;
					cubic = false;
					sample = tex3D(tex, pos.x, pos.y, pos.z);
					col = tex1D(transferTex, (sample-transferOffset)*transferScale);
					gradPos.x = pos.x;
					gradPos.y = pos.y;
					gradPos.z = pos.z;

					preValue = tex3D(tex, (gradPos.x-tstepGrad), gradPos.y, gradPos.z);
					postValue = tex3D(tex, (gradPos.x+tstepGrad), gradPos.y, gradPos.z);
					grad_x = (postValue-preValue)/(2.0f*tstepGrad);

					preValue = tex3D(tex, gradPos.x, (gradPos.y-tstepGrad), gradPos.z);
					postValue = tex3D(tex, gradPos.x, (gradPos.y+tstepGrad), gradPos.z);
					grad_y = (postValue-preValue)/(2.0f*tstepGrad);

					preValue = tex3D(tex, gradPos.x, gradPos.y, (gradPos.z-tstepGrad));
					postValue = tex3D(tex, gradPos.x, gradPos.y, (gradPos.z+tstepGrad));
					grad_z = (postValue-preValue)/(2.0f*tstepGrad);

					float3 dir = normalize(eyeRay.d);
					float3 norm = normalize(make_float3(grad_x, grad_y,grad_z));
					I_dif = max(dot(norm, dir))*kd;
					float3 R = normalize(dir + (2.0 * dot(dir,norm)*norm));
					float I_spec = pow(max(dot(dir, R)), powerFactor);
					phong = clamp(I_dif + I_spec+ ka * I_amb, 0.0, 1.0);
					col.w *= density;
					col.x = I_amb* col.w  + clamp(col.w*col.x*(phong), 0.0, 1.0);
					col.y = I_amb* col.w  + clamp(col.w*col.y*(phong), 0.0, 1.0);
					col.z = I_amb* col.w  + clamp(col.w*col.z*(phong), 0.0, 1.0);

					sum = sum + col*pow((1.0f - sum.w),(0.004f/tstep));

				}
				else if(isoSurface)
				{
					lightingCondition = false;
					cubic = false;
					float temp1, temp2;
					start = pos;
					next = pos + eyeRay.d*tstep;
					float3 coord1, coord2;
					coord1.x = start.x*x_dim;
					coord1.y = start.y*y_dim;
					coord1.z = start.z*z_dim;
					coord2.x = next.x*x_dim;
					coord2.y = next.y*y_dim;
					coord2.z = next.z*z_dim;
					if(isoLinear)
					{
						temp1 = tex3D(tex, start.x , start.y , start.z );
						temp2 = tex3D(tex, next.x , next.y , next.z );
					}
					else
					{
						temp1 = cubicTex3D(tex_cubic, coord1);
						temp2 = cubicTex3D(tex_cubic, coord2);
					}

					float val1 = temp1 - isoValue;
					float val2 = temp2 - isoValue;
					if(val1*val2<0)
					{
						value = bisection(start,next,eyeRay.d,tstep,isoValue);
						sample = value.w;
						gradPos.x = value.x;
						gradPos.y = value.y;
						gradPos.z = value.z;

						flag = true;
					}
					else if(val1 == isoValue)
					{
						sample = temp1;
						gradPos.x = start.x;
						gradPos.y = start.y;
						gradPos.z = start.z;
						flag = true;
					}
					else if(val2 == isoValue)
					{
						sample = temp2;
						gradPos.x = next.x;
						gradPos.y = next.y;
						gradPos.z = next.z;
						flag = true;
					}
					if(flag)
					{
						sum = tex1D(transferTexIso, (sample-transferOffset)*transferScale);
						preValue = tex3D(tex, (gradPos.x-tstepGrad) , gradPos.y , gradPos.z );
						postValue = tex3D(tex, (gradPos.x+tstepGrad) , gradPos.y , gradPos.z );
						grad_x = (postValue-preValue)/(2*tstepGrad);

						preValue = tex3D(tex, gradPos.x , (gradPos.y-tstepGrad) , gradPos.z );
						postValue = tex3D(tex, gradPos.x , (gradPos.y+tstepGrad) , gradPos.z );
						grad_y = (postValue-preValue)/(2*tstepGrad);

						preValue = tex3D(tex, gradPos.x , gradPos.y , (gradPos.z-tstepGrad) );
						postValue = tex3D(tex, gradPos.x , gradPos.y , (gradPos.z+tstepGrad) );
						grad_z = (postValue-preValue)/(2*tstepGrad);

						float3 dir = normalize(eyeRay.d);
						float3 norm = normalize(make_float3(grad_x, grad_y,grad_z));
						//norm = normalize(mul(c_invViewMatrix, norm));
						//I_dif = fabs(dot(norm, -eyeRay.d))*kd;
						I_dif = max(dot(norm, dir))*kd;
						float3 R = normalize(dir + (2.0 * dot(dir,norm)*norm));
						float I_spec = pow(max(dot(dir, R)), powerFactor);
						//phong = clamp(I_dif + I_spec+ ka * I_amb, 0.0, 1.0);
						phong = clamp(I_dif + ka * I_amb, 0.0, 1.0);
						sum.x = (173.0/255.0)* phong;
						sum.y = (142.0/255.0) *phong;
						sum.z = (116.0/255.0) *phong;
						sum.w = 1;
						break;
					}

				}
				else if(cubic)
				{
					isoSurface = false;
					lightingCondition = false;


					float3 coord;
					coord.x = pos.x*x_dim;
					coord.y = pos.y*y_dim;
					coord.z = pos.z*z_dim;
					if(filterMethod == 1){
						sample = linearTex3D(tex_cubic, coord);
					}
					else if(filterMethod == 2){
						sample = cubicTex3D(tex_cubic, coord);
					}
					else
					{
						sample = cubicTex3D(tex_cubic, coord);
					}
					col = tex1D(transferTex, (sample - transferOffset)*transferScale);

					if(cubicLight)
					{
						gradPos.x = pos.x;
						gradPos.y = pos.y;
						gradPos.z = pos.z;


						preValue = cubicTex3D(tex_cubic, ((gradPos.x-tstepGrad))*x_dim, (gradPos.y)*y_dim, (gradPos.z)*z_dim);
						postValue = cubicTex3D(tex_cubic, ((gradPos.x+tstepGrad))*x_dim, (gradPos.y)*y_dim, (gradPos.z)*z_dim);
						grad_x = (postValue-preValue)/(2.0f*tstepGrad*x_dim);

						preValue = cubicTex3D(tex_cubic, (gradPos.x)*x_dim, ((gradPos.y-tstepGrad))*y_dim, (gradPos.z)*z_dim);
						postValue = cubicTex3D(tex_cubic, (gradPos.x)*x_dim, ((gradPos.y+tstepGrad))*y_dim, (gradPos.z)*z_dim);
						grad_y = (postValue-preValue)/(2.0f*tstepGrad*y_dim);

						preValue = cubicTex3D(tex_cubic, (gradPos.x)*x_dim, (gradPos.y)*y_dim, ((gradPos.z-tstepGrad))*z_dim);
						postValue = cubicTex3D(tex_cubic, (gradPos.x)*x_dim, (gradPos.y)*y_dim, ((gradPos.z+tstepGrad))*z_dim);
						grad_z = (postValue-preValue)/(2.0f*tstepGrad*z_dim);
						float3 dir = normalize(eyeRay.d);
						float3 norm = normalize(make_float3(grad_x, grad_y,grad_z));
						I_dif = max(dot(norm, dir))*kd;
						float3 R = normalize(dir + (2.0 * dot(dir,norm)*norm));
						float I_spec = pow(max(dot(dir, R)), powerFactor);
						phong = clamp(I_dif + I_spec+ ka * I_amb, 0.0, 1.0);
						col.w *= density;
						col.x = I_amb* col.w  + clamp(col.w*col.x*(phong), 0.0, 1.0);
						col.y = I_amb* col.w  + clamp(col.w*col.y*(phong), 0.0, 1.0);
						col.z = I_amb* col.w  + clamp(col.w*col.z*(phong), 0.0, 1.0);

					}
					else
					{
						col.w *= density;
						col.x *= col.w;
						col.y *= col.w;
						col.z *= col.w;

					}

					sum = sum + col*pow((1.0f - sum.w), (0.004f/tstep));

				}
				else
				{
					sample = tex3D(tex, pos.x, pos.y, pos.z);
					col = tex1D(transferTex, (sample-transferOffset)*transferScale);
					col.w *= density;
					col.x *= col.w;
					col.y *= col.w;
					col.z *= col.w;
					sum = sum + col*pow((1.0f - sum.w),(0.004f/tstep));

				}

				if (sum.w > opacityThreshold)
				{
					break;
				}

				t += tstep;

				if (t > tfar) break;

				pos += step;
			}
			sum *= brightness;
			sum = sum + backGround * (1.0f - sum.w);

			accumulatedSum[i] = sum;
		}

	}
//	sum *= brightness;
	for(int i=0; i<noRays; i++)
	{
		finalSum += accumulatedSum[i];
	}
	averageSum = finalSum/float(noRays);



	d_red[tempLin] = averageSum.x;
	res_red[tempLin] = averageSum.x;
	d_green[tempLin] = averageSum.y;
	res_green[tempLin] = averageSum.y;
	d_blue[tempLin] = averageSum.z;
	res_blue[tempLin] = averageSum.z;
}



void render_kernel(dim3 gridSize, dim3 blockSize,int *d_pattern, int *linPattern, int *d_xPattern, int *d_yPattern, float *d_vol, float *d_red, float *d_green, float *d_blue,
		float *res_red, float *res_green, float *res_blue, float *device_x, float *device_p, int imageW, int imageH, float density, float brightness, float transferOffset,
		float transferScale,bool isoSurface, float isoValue, bool lightingCondition, bool isoLinear, float tstep, bool cubic, bool cubicLight, bool superSample, int filterMethod, float *d_temp)
{
	//	hipEventCreate(&start);
	//	hipEventRecord(start,0);
	d_render<<<gridSize, 256>>>(d_pattern, linPattern, d_xPattern, d_yPattern, d_vol, d_red, d_green, d_blue,res_red, res_green, res_blue,
			imageW, imageH, density, brightness, transferOffset, transferScale, isoSurface, isoValue, lightingCondition, isoLinear, tstep, cubic, cubicLight, superSample, filterMethod, d_temp);
	hipDeviceSynchronize();

}
//d_output, d_vol, res_red, res_green, res_blue, imageW, imageH, d_xPattern, d_yPattern, d_linear
__global__ void blend(bool reconstruction, uint *d_output,float *d_vol, float *res_red, float *res_green, float *res_blue, int imageW, int imageH, float *d_red, float *d_green, float *d_blue)
{

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	if((x>=imageW)||(y>=imageH))
		return;

	int index = x + y * imageW;

	float4 temp = make_float4(0.0f);

	if(reconstruction)
	{
		temp.x = res_red[index];
		temp.y = res_green[index];
		temp.z = res_blue[index];
	}
	else
	{
		temp.x = d_red[index];
		temp.y = d_green[index];
		temp.z = d_blue[index];

	}

	d_output[index] = rgbaFloatToInt(temp);



}
//    blendFunction(gridVol, blockSize, d_output,d_vol, res_red, res_green, res_blue, height, width, d_xPattern, d_yPattern, d_linear);
void blendFunction(dim3 grid, dim3 block, bool reconstruction, uint *d_output, float *d_vol, float *res_red, float *res_green, float *res_blue, int imageH, int imageW, float *d_red, float *d_green, float *d_blue)
{
	//	 blend<<<grid, block>>>(d_output, res_red, res_green, res_blue, imageW, imageH);
	blend<<<grid, block>>>(reconstruction, d_output, d_vol, res_red, res_green, res_blue, imageW, imageH, d_red, d_green, d_blue);
}

/*
void reconstructionFunction(dim3 grid, dim3 block, float *data, float *red, float *green, float *blue,
 		int *pattern, float *kernel, float *d_result,float *red_res, float *green_res, float *blue_res,
 		int maskH, int maskW, int dataH, int dataW, float *device_x, float *device_p)
 */


void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

