#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "helper_functions.h"
#include "CI/memcpy.cu"
#include "CI/cubicPrefilter3D.cu"
#include "CI/cubicTex3D.cu"

#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "deviceVars.h"
#include "reconstruction.h"

#define PAD 3

hipEvent_t start, stop;
float volumeTime;

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
//typedef unsigned short VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;         // 3D texture
texture<float4, 1, hipReadModeElementType>         transferTex; // 1D transfer function texture
texture<float4, 1, hipReadModeElementType>         transferTexIso;
texture<uchar, 3, hipReadModeNormalizedFloat> tex_cubic;
texture<float, 3, hipReadModeElementType> coeffs;


typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__ float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__ float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}


void setTextureFilterMode(bool bLinearFilter)
{
    tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

void initCuda(void *h_volume, hipExtent volumeSize)
{
    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
    checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // set texture parameters
    tex.normalized = true;                      // access with normalized texture coordinates
    tex.filterMode = hipFilterModeLinear;      // linear interpolation
    tex.addressMode[0] = hipAddressModeBorder;  // clamp texture coordinates //hipAddressModeClamp
    tex.addressMode[1] = hipAddressModeBorder;
    tex.addressMode[2] = hipAddressModeBorder;
    // bind array to 3D texture
    checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
/*
    // create transfer function texture
    float4 transferFunc[] =
    {
        {  0.0, 0.0, 0.0, 0.0, },
        {  1.0, 0.0, 0.0, 1.0, },
        {  1.0, 0.5, 0.0, 1.0, },
        {  1.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 0.0, 1.0, },
        {  0.0, 1.0, 1.0, 1.0, },
        {  0.0, 0.0, 1.0, 1.0, },
        {  1.0, 0.0, 1.0, 1.0, },
        {  0.0, 0.0, 0.0, 0.0, },
    };
*/
    float4 transferFunc[] =
	{
		{0.231372549,	0.298039216,	0.752941176,	0,},
		{0.266666667,	0.352941176,	0.8,	0.03125,},
		{0.301960784,	0.407843137,	0.843137255,	0.0625,},
		{0.341176471,	0.458823529,	0.882352941,	0.09375,},
		{0.384313725,	0.509803922,	0.917647059,	0.125,},
		{0.423529412,	0.556862745,	0.945098039,	0.15625,},
		{0.466666667,	0.603921569,	0.968627451,	0.1875,},
		{0.509803922,	0.647058824,	0.984313725,	0.21875,},
		{0.552941176,	0.690196078,	0.996078431,	0.25,},
		{0.596078431,	0.725490196,	1,	0.28125,},
		{0.639215686,	0.760784314,	1,	0.3125,},
		{0.682352941,	0.788235294,	0.992156863,	0.34375,},
		{0.721568627,	0.815686275,	0.976470588,	0.375,},
		{0.760784314,	0.835294118,	0.956862745,	0.40625,},
		{0.8,	0.850980392,	0.933333333,	0.4375,},
		{0.835294118,	0.858823529,	0.901960784,	0.46875,},
		{0.866666667,	0.866666667,	0.866666667,	0.5,},
		{0.898039216,	0.847058824,	0.819607843,	0.53125,},
		{0.925490196,	0.82745098,	0.77254902,	0.5625,},
		{0.945098039,	0.8,	0.725490196,	0.59375,},
		{0.960784314,	0.768627451,	0.678431373,	0.625,},
		{0.968627451,	0.733333333,	0.62745098,	0.65625,},
		{0.968627451,	0.694117647,	0.580392157,	0.6875,},
		{0.968627451,	0.650980392,	0.529411765,	0.71875,},
		{0.956862745,	0.603921569,	0.482352941,	0.75,},
		{0.945098039,	0.552941176,	0.435294118,	0.78125,},
		{0.925490196,	0.498039216,	0.388235294,	0.8125,},
		{0.898039216,	0.439215686,	0.345098039,	0.84375,},
		{0.870588235,	0.376470588,	0.301960784,	0.875,},
		{0.835294118,	0.31372549,	0.258823529,	0.90625,},
		{0.796078431,	0.243137255,	0.219607843,	0.9375,},
		{0.752941176,	0.156862745,	0.184313725,	0.96875,},
		{0.705882353,	0.015686275,	0.149019608,	1,}
	};
    hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
    hipArray *d_transferFuncArray;
    checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc)/sizeof(float4), 1));
    checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    checkCudaErrors(hipBindTextureToArray(transferTex, d_transferFuncArray, channelDesc2));

    //Creating TransferTexIso
    float4 transferFuncIso[] =
	{
//		{  0.0, 0.0, 0.0, 0.0, },
		{  1.0, 1.0, 1.0, 1.0, }
	};

	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArrayIso;
	checkCudaErrors(hipMallocArray(&d_transferFuncArrayIso, &channelDesc3, sizeof(transferFuncIso)/sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArrayIso, 0, 0, transferFuncIso, sizeof(transferFuncIso), hipMemcpyHostToDevice));

	transferTexIso.filterMode = hipFilterModeLinear;
	transferTexIso.normalized = true;    // access with normalized texture coordinates
	transferTexIso.addressMode[0] = hipAddressModeBorder;   // wrap texture coordinates

	// Bind the array to the texture
	checkCudaErrors(hipBindTextureToArray(transferTexIso, d_transferFuncArrayIso, channelDesc3));





}

void freeCudaBuffers()
{
    checkCudaErrors(hipFreeArray(d_volumeArray));
    checkCudaErrors(hipFreeArray(d_transferFuncArray));
}

void initCudaCubicSurface(const uchar* voxels, uint3 volumeSize)
{

    // calculate the b-spline coefficients
    hipPitchedPtr bsplineCoeffs = CastVolumeHostToDevice(voxels, volumeSize.x, volumeSize.y, volumeSize.z);
    CubicBSplinePrefilter3DTimer((float*)bsplineCoeffs.ptr, (uint)bsplineCoeffs.pitch, volumeSize.x, volumeSize.y, volumeSize.z);

    // create the b-spline coefficients texture
    hipArray *coeffArray = 0;
    hipExtent volumeExtent = make_hipExtent(volumeSize.x, volumeSize.y, volumeSize.z);
    CreateTextureFromVolume(&coeffs, &coeffArray, bsplineCoeffs, volumeExtent, true);
//    CUDA_SAFE_CALL(hipFree(bsplineCoeffs.ptr));  //they are now in the coeffs texture, we do not need this anymore
	hipFree(bsplineCoeffs.ptr);
    // Now create a texture with the original sample values for nearest neighbor and linear interpolation
    // Note that if you are going to do cubic interpolation only, you can remove the following code

    CreateTextureFromVolume(&tex_cubic, &volumeArray, voxels, volumeExtent, false);
    tex_cubic.addressMode[0] = hipAddressModeBorder;
    tex_cubic.addressMode[1] = hipAddressModeBorder;
    tex_cubic.addressMode[2] = hipAddressModeBorder;




}


__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

__device__ float4 bisection(float3 start, float3 next,float3 direction, float stepSize, float isoValue)
{
    float tstep = stepSize/2;
    float3 a= start;
    float3 b = start+direction*tstep;
    float3 c = next;
    float3 point;
    float val = 0.0f;
    float temp_a = tex3D(tex, a.x , a.y , a.z ) - isoValue;
    float temp_b = tex3D(tex, b.x , b.y , b.z ) - isoValue;
    float temp_c = tex3D(tex, c.x , c.y , c.z ) - isoValue;
    int count = 0;
    float4 sample = make_float4(0.0f);

    while(count<25)
    {

        if(fabs(temp_b) <= (1e-6))
        {
            break;
        }

        if(temp_a*temp_b < 0)
        {
            tstep = tstep/2;
            c = b;
            b = a + direction * tstep;
        }
        else if(temp_b * temp_c < 0)
        {
            a = b;
            tstep = (3/4)*stepSize;
            b = a + direction*tstep;
        }
        val = tex3D(tex, b.x , b.y , b.z );
        point = b;
        if(fabs(val - isoValue)<= (1e-6))
        {
            break;
        }
        count++;
    }

 /*
         while(count<25)
    {
        if(fabs(temp_b) <= (1e-6))
        {
            break;
        }
        if(temp_a*temp_b < 0)
        {
            tstep = tstep/2;
            c = b;
            b = a + direction * tstep;
        }
        else if(temp_b * temp_c < 0)
        {
            a = b;
            tstep = (3/4)*stepSize;
            b = a + direction*tstep;
        }
        val = tex3D(tex, b.x , b.y , b.z );
        point = b;
        if(fabs(val - isoValue)<= (1e-6))
        {
            break;
        }
        count++;
    }
   */
    sample.w = val;
    sample.x = b.x;
    sample.y = b.y;
    sample.z = b.z;

    return sample;
}



__global__ void d_render(int *d_pattern, int *d_xPattern, int *d_yPattern, float *d_red, float *d_green, float *d_blue, float *d_opacity, float *res_red, float *res_green, float *res_blue, float *res_opacity, int imageW, int imageH,
					float density, float brightness,float transferOffset, float transferScale, bool isoSurface, float isoValue, bool lightingCondition)
{
    const int maxSteps =1000;
    const float tstep = 0.001f;
    const float opacityThreshold = 0.95f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);
    float4 sum, col;
	float ka = 0.3f;
	float I_amb = 0.3;
	float kd = 0.5;
	float I_dif;
	float ks = 0.5;
	float I_spec;
	float phong = 0.0f;
	float tstepGrad = 0.00009f;
	float4 value;
	float sample;


    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    int index = int(x) + int(y) * imageW;
/*
    if(d_pattern[index] == 0)
    {
    	return;
    }
*/

    if ((x >= imageW) || (y >= imageH))
    	return;

    float u = ((x+0.5f) / (float) imageW)*2.0f-1.0f;
    float v = ((y+0.5f) / (float) imageH)*2.0f-1.0f;
/*
    float u = (d_xPattern[index]/(float)imageW)*2.0f - 1.0f;
    float v = (d_yPattern[index]/(float)imageH)*2.0f - 1.0f;
    */
    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit)
    	{
 //   	sum = make_float4(0.0f);

    	d_red[index] = 0.0f;
		res_red[index] = 0.0f;
		d_green[index] = 0.0f;
		res_green[index] = 0.0f;
		d_blue[index] = 0.0f;
		res_blue[index] = 0.0f;

		return;

    	}
    else
    {

    	float grad_x, grad_y, grad_z;


    	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
    	sum = make_float4(0.0f);
		// march along ray from front to back, accumulating color
		float t = tnear;
		float3 pos = eyeRay.o + eyeRay.d*tnear;
		float3 step = eyeRay.d*tstep;
		col = make_float4(0.0f);
		sample = 0.0f;
		float3 next;
		float3 start, mid, end, gradPos;
		float preValue, postValue;
//		bool lightCondition = true;
//		bool isoSurface = false  ;
		bool cubic;// = false; true
		bool flag = false;

		pos.x = (pos.x *0.5f + 0.5f);//*(x_dim/x_dim)*(x_space/x_space); //pos.x = (pos.x *0.5f + 0.5f)/x_aspect;
		pos.y = (pos.y *0.5f + 0.5f);//(x_dim/y_dim)*(x_space/x_space);
		pos.z = (pos.z *0.5f + 0.5f);//(x_dim/z_dim)*(x_space/z_space);
		/*
		pos.x = (pos.x *0.5f + 0.5f)/x_aspect;//*(x_dim/x_dim)*(x_space/x_space); //pos.x = (pos.x *0.5f + 0.5f)/x_aspect;
		pos.y = (pos.y *0.5f + 0.5f)/y_aspect;//(x_dim/y_dim)*(x_space/x_space);
		pos.z = (pos.z *0.5f + 0.5f);//(x_dim/z_dim)*(x_space/z_space);
		*/
		for (int i=0; i<maxSteps; i++)
		{


			if(lightingCondition)
			{
				isoSurface = false;
				sample = tex3D(tex, pos.x, pos.y, pos.z);
				col = tex1D(transferTex, (sample-transferOffset)*transferScale);
				gradPos.x = pos.x;
				gradPos.y = pos.y;
				gradPos.z = pos.z;

				preValue = tex3D(tex, (gradPos.x-tstepGrad), gradPos.y, gradPos.z);
				postValue = tex3D(tex, (gradPos.x+tstepGrad), gradPos.y, gradPos.z);
				grad_x = (postValue-preValue)/2.0f*tstepGrad;

				preValue = tex3D(tex, gradPos.x, (gradPos.y-tstepGrad), gradPos.z);
				postValue = tex3D(tex, gradPos.x, (gradPos.y+tstepGrad), gradPos.z);
				grad_y = (postValue-preValue)/2.0f*tstepGrad;

				preValue = tex3D(tex, gradPos.x, gradPos.y, (gradPos.z-tstepGrad));
				postValue = tex3D(tex, gradPos.x, gradPos.y, (gradPos.z+tstepGrad));
				grad_z = (postValue-preValue)/2.0f*tstepGrad;

				float3 dir = normalize(-eyeRay.d);
				float3 norm = normalize(make_float3(grad_x, grad_y,grad_z));


				I_dif = fabs(dot(norm, dir))*1.0f;

				float3 R = dir + (2.0f * norm * kd);
				I_spec = pow(dot(dir, R)*ks, 30.0f);

				phong = I_dif + I_spec + ka * I_amb;


				col.w *= density;

				col.x = I_amb* col.w  + clamp(col.w*col.x*(phong), 0.0, 1.0);
				col.y = I_amb* col.w  + clamp(col.w*col.y*(phong), 0.0, 1.0);
				col.z = I_amb* col.w  + clamp(col.w*col.z*(phong), 0.0, 1.0);

			}
			else if(isoSurface)
			{
				lightingCondition = false;
//				cubic = highQuality;
				start = pos;
				next = pos + eyeRay.d*tstep;
				float temp1 = tex3D(tex, start.x , start.y , start.z );
				float temp2 = tex3D(tex, next.x , next.y , next.z );

				float val1 = temp1 - isoValue;
				float val2 = temp2 - isoValue;
				if(val1*val2<0)
				{
					value = bisection(start,next,eyeRay.d,tstep,isoValue);
					sample = value.w;
					gradPos.x = value.x;
					gradPos.y = value.y;
					gradPos.z = value.z;

					flag = true;
				}
				else if(val1 == isoValue)
				{
					sample = temp1;
					gradPos.x = start.x;
					gradPos.y = start.y;
					gradPos.z = start.z;
					flag = true;
				}
				else if(val2 == isoValue)
				{
					sample = temp2;
					gradPos.x = next.x;
					gradPos.y = next.y;
					gradPos.z = next.z;
					flag = true;
				}
				if(flag)
				{
					sum = tex1D(transferTexIso, (sample-transferOffset)*transferScale);
//					col = tex1D(transferTexIso, (sample-transferOffset)*transferScale);
					col = make_float4(1.0f);

					preValue = tex3D(tex, (gradPos.x-tstepGrad) , gradPos.y , gradPos.z );
					postValue = tex3D(tex, (gradPos.x+tstepGrad) , gradPos.y , gradPos.z );
					grad_x = (postValue-preValue)/2*tstepGrad;

					preValue = tex3D(tex, gradPos.x , (gradPos.y-tstepGrad) , gradPos.z );
					postValue = tex3D(tex, gradPos.x , (gradPos.y+tstepGrad) , gradPos.z );
					grad_y = (postValue-preValue)/2*tstepGrad;

					preValue = tex3D(tex, gradPos.x , gradPos.y , (gradPos.z-tstepGrad) );
					postValue = tex3D(tex, gradPos.x , gradPos.y , (gradPos.z+tstepGrad) );
					grad_z = (postValue-preValue)/2*tstepGrad;

					float3 norm = normalize(make_float3(grad_x, grad_y,grad_z));
					norm = normalize(mul(c_invViewMatrix, norm));
					I_dif = fabs(dot(norm, -eyeRay.d))*kd;
					float3 R = -eyeRay.d + (2 * norm * kd);
					float I_spec = pow(dot(-eyeRay.d, R)*ks,0.05f);

					phong = kd*I_dif+I_spec*ks;
					sum.x = sum.x*phong;
					sum.y = sum.y*phong;
					sum.z = sum.z*phong;
					sum.w = 1;
					break;
				}
				else
					col= make_float4(0.0f);


			}
			else
			{

				sample = tex3D(tex, pos.x, pos.y, pos.z);
				col = tex1D(transferTex, (sample-transferOffset)*transferScale);
				col.w *= density;

				// "under" operator for back-to-front blending
				//sum = lerp(sum, col, col.w);

				// pre-multiply alpha
				col.x *= col.w;
				col.y *= col.w;
				col.z *= col.w;
			}



			// "over" operator for front-to-back blending
//			sum = sum + col*(1.0f - sum.w);
			sum = sum + col*pow((1.0f - sum.w),(0.004f/tstep));

			// exit early if opaque
			if (sum.w > opacityThreshold)
				break;

			t += tstep;

			if (t > tfar) break;

			pos += step;
		}

		sum *= brightness;

		d_red[index] = sum.x;
		res_red[index] = sum.x;
		d_green[index] = sum.y;
		res_green[index] = sum.y;
		d_blue[index] = sum.z;
		res_blue[index] = sum.z;
    }

}

__global__ void blend(uint *d_output, float *res_red, float *res_green, float *res_blue, int imageW, int imageH)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	int index = x + y * imageW;

	if ((x >= imageW) || (y >= imageH))
	    	return;
	float4 temp = make_float4(0.0f);
//	d_output[index] = rgbaFloatToInt(temp);
//	temp.w = res_opacity[index];
	temp.x = res_red[index];
	temp.y = res_green[index];
	temp.z = res_blue[index];
	d_output[index] = rgbaFloatToInt(temp);
//	d_output[index] = rgbaFloatToInt(make_float4(res_red[index], res_green[index], res_blue[index], res_opacity[index]));

}

void render_kernel(dim3 gridSize, dim3 blockSize, int *d_pattern, int *d_xPattern, int *d_yPattern, float *d_red, float *d_green, float *d_blue,
		float *d_opacity, float *res_red, float *res_green, float *res_blue, float *res_opacity, float *device_x, float *device_p, int imageW, int imageH,
		float density, float brightness, float transferOffset, float transferScale,bool isoSurface, float isoValue, bool lightingCondition)
{
//	hipEventCreate(&start);
//	hipEventRecord(start,0);
	 d_render<<<gridSize, blockSize>>>(d_pattern, d_xPattern, d_yPattern, d_red, d_green, d_blue,d_opacity, res_red, res_green, res_blue, res_opacity,
	    		imageW, imageH, density, brightness, transferOffset, transferScale, isoSurface, isoValue, lightingCondition);
    hipDeviceSynchronize();
    /*
    d_render<<<gridSize, blockSize>>>(d_pattern, d_xPattern, d_yPattern, d_red, d_green, d_blue,d_opacity, res_red, res_green, res_blue, res_opacity,
    		imageW, imageH, density, brightness, transferOffset, transferScale);
     */


//    hipEventCreate(&stop);
//    hipEventRecord(stop, 0);
//    hipEventElapsedTime(&volumeTime, start, stop);
//    printf("Volume time: %f ms\n", volumeTime);

 //   blend<<<gridSize, blockSize>>>(d_output, d_red, d_green, d_blue, d_opacity, imageW, imageH);

//    hipDeviceSynchronize();
    //    reconstructionKernel<<<grid,block>>>(data, d_result, pattern, dataH, dataW, device_x, device_p);


//    reconstructionKernel<<<gridSize, blockSize>>>(d_red, res_red, d_pattern, imageH, imageW, device_x, device_p);
//    reconstructionKernel<<<gridSize, blockSize>>>(d_green, res_green, d_pattern, imageH, imageW, device_x, device_p);
//    reconstructionKernel<<<gridSize, blockSize>>>(d_blue, res_blue, d_pattern, imageH, imageW, device_x, device_p);
//    reconstructionKernel<<<gridSize, blockSize>>>(d_opacity, res_opacity, d_pattern, imageH, imageW, device_x, device_p);
//    blend<<<gridSize, blockSize>>>(d_output, d_red, d_green, d_blue, d_opacity, imageW, imageH);

//    reconstructionFunction(gridSize, blockSize, d_red, d_green, d_blue, d_pattern, res_red, res_green, res_blue, imageH, imageW, device_x, device_p);


}

void blendFunction(dim3 grid, dim3 block,uint *d_output, float *res_red, float *res_green, float *res_blue, int imageH, int imageW)
{
	 blend<<<grid, block>>>(d_output, res_red, res_green, res_blue, imageW, imageH);
}

/*
void reconstructionFunction(dim3 grid, dim3 block, float *data, float *red, float *green, float *blue,
 		int *pattern, float *kernel, float *d_result,float *red_res, float *green_res, float *blue_res,
 		int maskH, int maskW, int dataH, int dataW, float *device_x, float *device_p)
*/


void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}
